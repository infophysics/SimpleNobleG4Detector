#include "hip/hip_runtime.h"
// C++ includes.
#include <iostream>
#include <vector>

// CUDA includes.
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Custom includes.
#include "EvalCUDA.hh"

#define NBINS 100
#define BINL 0.5
#define NEVENTS 5000

namespace ReCUDA
{
	__global__ void GetR( double *dQBins, double *dEBins, double *R )
	{
		int Index = NBINS * threadIdx.x + blockIdx.x;
		R[ Index ] = dQBins[ Index ] / dEBins[ Index ];
	}
	
	__global__ void Birks( 
		double 	*dE, 
		double 	*dx, 
		int 	*B, 
		double 	F, 
		double* p, 
		size_t 	n, 
		double 	*dQBins, 
		double 	*dEBins 
	)
	{
		int ThreadID = blockIdx.x * blockDim.x + threadIdx.x;
		int ParameterID = 2 * threadIdx.y;
		double StepdQ, StepdE;
		
		if( ThreadID < n )
		{
			StepdQ = ( p[ParameterID + 0] * ( dE[ThreadID] / dx[ThreadID] ) /
				( 1 + ( p[ParameterID + 1] / F ) *
				( dE[ThreadID] / dx[ThreadID] ) ) ) * dx[ThreadID];
			StepdE = dE[ThreadID];
		
			if( !std::isnan(StepdQ) ) {
				atomicAdd(dQBins + NBINS * threadIdx.y + B[ThreadID], StepdQ / (BINL * NEVENTS));
			}
			if( !std::isnan(StepdE) ) {
				atomicAdd(dEBins + NBINS * threadIdx.y + B[ThreadID], StepdE / (BINL * NEVENTS));
			}
		}
	}

	__global__ void DMBirks( double *dE, double *dx, double F, size_t n, double* p, double* QY )
	{
		int ThreadID = blockIdx.x * blockDim.x + threadIdx.x;
		int ParameterID = 2 * threadIdx.y;
		double StepdQ;
		
		if( ThreadID < n )
		{
			StepdQ = ( p[ParameterID + 0] * ( dE[ThreadID] / dx[ThreadID] ) /
				( 1 + ( p[ParameterID + 1] / F ) *
				( dE[ThreadID] / dx[ThreadID] ) ) ) * dx[ThreadID];
			if( !std::isnan(StepdQ) ) {
				atomicAdd( QY + threadIdx.y, StepdQ / ( 1000 ) );
			}
		}
	}

	void DMBirksWrapper(
		const std::vector<double> &dE, 
		const std::vector<double> &dx,
		const std::vector<double> &F, 
		const std::vector<double> &p,
		std::vector< std::vector<double> > &QY 
	)
	{
		// Set the thread-dimensions of each block.
		const int NThreadEntries(32);
		const int NThreadParameters(32);
		const dim3 ThreadDimensions( NThreadEntries, NThreadParameters );
		
		// We will need a few constants/dimensions for the calculation.
		const size_t N(dE.size());
		const size_t NFields(F.size());

		// Create the pointers for the arrays in device memory.
		double *DeviceG4dE;
		double *DeviceG4dx;
		double *DeviceP;
		double *DeviceQY;

		// Now we need to allocate memory on the device for these arrays. The raw input only needs
		// to be copied once, but the parameters will only be processed in batches of 32 (allocate
		// memory in loop).

		hipMalloc( (void**)&DeviceG4dE, sizeof(double) * N );
		hipMalloc( (void**)&DeviceG4dx, sizeof(double) * N );
		hipMalloc( (void**)&DeviceP, sizeof(double) * 64 );
		hipMalloc( (void**)&DeviceQY, sizeof(double) * 32 );

		// Now copy the raw G4 input to the device.
		hipMemcpy( DeviceG4dE, &dE[0], sizeof(double) * N, hipMemcpyHostToDevice );
		hipMemcpy( DeviceG4dx, &dx[0], sizeof(double) * N, hipMemcpyHostToDevice );

		// So the major time cost with outsourcing the recombination calculation to a GPU is in
		// copying to and from memory, and other overhead. We can minimize this by copying the
		// G4 input to the device memory once and performing all calculations on it at once. We will
		// need to copy the dQ calculations back to the host and the batch of parameters to the
		// device, but this is relatively little compared to the large size of the raw input.

		size_t ParHead;
		for( size_t fi(0); fi < NFields; ++fi )
		{
		
			ParHead = 0;
			while( ParHead + 64 <= p.size() )
			{
				// Reset the arrays allocated for the QY calculation.
				hipMemset( DeviceQY, 0, sizeof(double) * 32 );
				// Copy the current batch of parameters to the device.
				hipMemcpy( DeviceP, &p[ParHead], sizeof(double) * 64, hipMemcpyHostToDevice );
				// Make a kernel call for this batch of parameters and field.
				DMBirks <<< ( N / NThreadEntries + 1 ), ThreadDimensions >>>( DeviceG4dE,
												DeviceG4dx,
												F[fi],
												N,
												DeviceP,
												DeviceQY );
				hipDeviceSynchronize();
				// Retrieve any error states (if any).
				hipError_t Error = hipGetLastError();
				if( Error != hipSuccess ) {
					std::cerr << "CUDA error: " << hipGetErrorString(Error) << std::endl;
				}	
				
				// Copy the resulting calculated values back to the host.
				hipMemcpy( &QY[fi][ 32 * (ParHead / 64) ], DeviceQY, sizeof(double) * 32, hipMemcpyDeviceToHost );
				ParHead += 64;
			}
		}
		
		// Once we are finished we need to release the memory that was allocated on the device.
		hipFree( DeviceG4dE );
		hipFree( DeviceG4dx );
		hipFree( DeviceP );
		hipFree( DeviceQY );
		
	}
	
	void BirksWrapper(
		const std::vector<double> &dE, 
		const std::vector<double> &dx,
		const std::vector<int> &B, 
		const std::vector<double> &F,
		const std::vector<double> &p, 
		std::vector<std::vector<double>>  &RBins,
		std::vector<std::vector<double>>  &dEBins 
	)
	{
		// Set the thread-dimensions of each block.
		const int NThreadEntries(32);
		const int NThreadParameters(32);
		const dim3 ThreadDimensions( NThreadEntries, NThreadParameters );
		
		// We will need a few constants/dimensions for the calculation.
		const size_t N(dE.size());
		const size_t NFields(F.size());
		
		// Create the pointers for the arrays in device memory.
		double *DeviceG4dE;
		double *DeviceG4dx;
		double *DevicedQBins;
		double *DevicedEBins;
		double *DeviceRBins;
		double *DeviceP;
		int *DeviceB;

		// Now we need to allocate memory on the device for these arrays. The raw input only needs
		// to be copied once, but the parameters will only be processed in batches of 32 (allocate
		// memory in loop).
		hipMalloc( (void**)&DeviceG4dE, sizeof(double) * N );
		hipMalloc( (void**)&DeviceG4dx, sizeof(double) * N );
		hipMalloc( (void**)&DevicedQBins, sizeof(double) * 32 * NBINS );
		hipMalloc( (void**)&DevicedEBins, sizeof(double) * 32 * NBINS );
		hipMalloc( (void**)&DeviceRBins, sizeof(double) * 32 * NBINS );
		hipMalloc( (void**)&DeviceP, sizeof(double) *  64 );
		hipMalloc( (void**)&DeviceB, sizeof(int) * N );

		// Now copy the raw G4 input to the device.
		hipMemcpy( DeviceG4dE, &dE[0], sizeof(double) * N, hipMemcpyHostToDevice );
		hipMemcpy( DeviceG4dx, &dx[0], sizeof(double) * N, hipMemcpyHostToDevice );
		hipMemcpy( DeviceB, &B[0], sizeof(int) * N, hipMemcpyHostToDevice );
		
		// So the major time cost with outsourcing the recombination calculation to a GPU is in
		// copying to and from memory, and other overhead. We can minimize this by copying the
		// G4 input to the device memory once and performing all calculations on it at once. We will
		// need to copy the dQ calculations back to the host and the batch of parameters to the
		// device, but this is relatively little compared to the large size of the raw input.
		
		size_t ParHead;
		for( size_t fi(0); fi < NFields; ++fi )
		{
			ParHead = 0;
			while( ParHead + 64 <= p.size() )
			{
				// Reset the arrays allocated for the segment dQ and dE calculation.
				hipMemset( DevicedQBins, 0, sizeof(double) * 32 * NBINS );
				hipMemset( DevicedEBins, 0, sizeof(double) * 32 * NBINS );
				// Copy the current batch of parameters to the device.
				hipMemcpy( DeviceP, &p[ParHead], sizeof(double) * 64, hipMemcpyHostToDevice );
				// Make a kernel call for this batch of parameters and field.
				Birks <<< ( N / NThreadEntries + 1 ), ThreadDimensions >>>( DeviceG4dE,
											DeviceG4dx,
											DeviceB,
											F[fi],
											DeviceP,
											N,
											DevicedQBins,
											DevicedEBins );
				hipDeviceSynchronize();
				// Retrieve any error states (if any).
				hipError_t Error = hipGetLastError();
				if( Error != hipSuccess ) {
					std::cerr << "CUDA error: " << hipGetErrorString(Error) << std::endl;
				}
				// Copy the resulting calculated values back to the host.
				GetR<<< NBINS, 32 >>>( DevicedQBins, DevicedEBins, DeviceRBins );
				hipMemcpy( &RBins[fi][ 32 * NBINS * (ParHead / 64) ], DeviceRBins, sizeof(double) * 32 * NBINS, hipMemcpyDeviceToHost );
				hipMemcpy( &dEBins[fi][ 32 * NBINS * (ParHead / 64) ], DevicedEBins, sizeof(double) * 32 * NBINS, hipMemcpyDeviceToHost );
				ParHead += 64;
			}
		}
		// Once we are finished we need to release the memory that was allocated on the device.
		hipFree( DeviceG4dE );
		hipFree( DeviceG4dx );
		hipFree( DevicedQBins );
		hipFree( DevicedEBins );
		hipFree( DeviceRBins );
		hipFree( DeviceP );
		hipFree( DeviceB );
	}
}
